#include <iostream>
#include <vector>
#include <queue>
#include <hip/hip_runtime.h>

__global__ void breadthFirstSearch(const int* graph, bool* visited, int startNode, int numNodes, double* executionTime) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid == startNode) {
        visited[tid] = true;
        printf("Visiting node %d\n", tid);


        for (int i = 0; i < numNodes; ++i) {
            if (graph[tid * numNodes + i] == 1 && !visited[i]) {
                visited[i] = true;
                printf("Visiting node %d\n", i);
            }
        }
    }
    __syncthreads();

    if (tid == 0) {
        *executionTime = clock() / static_cast<double>(CLOCKS_PER_SEC);
    }
}

int main() {
    int numNodes = 6;
    std::vector<int> graph = {
        0, 1, 1, 0, 0, 0,
        1, 0, 0, 1, 0, 0,
        1, 0, 0, 1, 1, 0,
        0, 1, 1, 0, 1, 1,
        0, 0, 1, 1, 0, 0,
        0, 0, 0, 1, 0, 0
    };

    int startNode = 0;

    int* deviceGraph;
    bool* deviceVisited;
    double* deviceExecutionTime;

    hipMalloc((void**)&deviceGraph, numNodes * numNodes * sizeof(int));
    hipMemcpy(deviceGraph, graph.data(), numNodes * numNodes * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceVisited, numNodes * sizeof(bool));
    hipMemset(deviceVisited, false, numNodes * sizeof(bool));

    hipMalloc((void**)&deviceExecutionTime, sizeof(double));
    hipMemset(deviceExecutionTime, 0, sizeof(double));

    breadthFirstSearch<<<1, numNodes>>>(deviceGraph, deviceVisited, startNode, numNodes, deviceExecutionTime);

    double executionTime;
    hipMemcpy(&executionTime, deviceExecutionTime, sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Execution Time: " << executionTime << " seconds" << std::endl;

    hipFree(deviceGraph);
    hipFree(deviceVisited);
    hipFree(deviceExecutionTime);

    return 0;
}
