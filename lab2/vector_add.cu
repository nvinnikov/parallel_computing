//#define N 10000000
#define MAX_INT 2147483647
#define SCALE 10000

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



__global__ void vector_add(float *out, float *a, float *b, int n) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if (i<n) {
        out[i]  = a[i] + b[i];
    }
}


int main() {
    int N = 1000000000;
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    float time;
    hipEvent_t start, stop;

    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    out = (float*)malloc(sizeof(float)*N);

    for (int i = 0; i < N; i++) {
        float base = ((float)rand())/MAX_INT;
        a[i] = SCALE*base;
        b[i] = a[i]/5.5;
    }

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_out, sizeof(float)*N);

    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    vector_add <<<(N/128)+1,128>>> (d_out, d_a, d_b, N); //16,32

    hipEventRecord(stop, 0);


    hipMemcpy(out, d_out, sizeof(float)*N, hipMemcpyDeviceToHost);

    hipEventElapsedTime(&time, start, stop);

    printf("Time spent executing by the GPU: %5.5f ms \n", time);
    printf("First element is: %5.5f\n", out[0]);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);

    return 0;


}
